#include <hip/hip_runtime.h>
#include <mma.h>
#include <hip/hip_fp16.h>
#include <iostream>
#include <vector>
#include <cassert>

using namespace nvcuda;

// Tensor Core FP16 accumulation test
__global__ void tensorcore_fp16_accumulation_test(
    const half* __restrict__ a,
    const half* __restrict__ b,
    half* __restrict__ c_fp16,
    float* __restrict__ c_fp32,
    int M, int N, int K) {
    
    // WMMA fragments
    wmma::fragment<wmma::matrix_a, 16, 16, 16, half, wmma::row_major> a_frag;
    wmma::fragment<wmma::matrix_b, 16, 16, 16, half, wmma::col_major> b_frag;
    wmma::fragment<wmma::accumulator, 16, 16, 16, half> c_frag_fp16;  // FP16 accumulation
    wmma::fragment<wmma::accumulator, 16, 16, 16, float> c_frag_fp32; // FP32 accumulation

    int warpM = (blockIdx.x * blockDim.x + threadIdx.x) / warpSize;
    int warpN = (blockIdx.y * blockDim.y + threadIdx.y);
    
    // Bounds check
    if (warpM * 16 >= M || warpN * 16 >= N) return;

    // Initialize accumulator fragments to zero
    wmma::fill_fragment(c_frag_fp16, __float2half(0.0f));
    wmma::fill_fragment(c_frag_fp32, 0.0f);

    // Perform matrix multiplication with both accumulation types
    for (int i = 0; i < K; i += 16) {
        int aRow = warpM * 16;
        int aCol = i;
        int bRow = i;
        int bCol = warpN * 16;
        
        // Bounds check for K dimension
        if (aCol + 16 <= K && bRow + 16 <= K) {
            // Load the inputs
            wmma::load_matrix_sync(a_frag, a + aRow * K + aCol, K);
            wmma::load_matrix_sync(b_frag, b + bRow * N + bCol, N);
            
            // Perform the matrix operations
            wmma::mma_sync(c_frag_fp16, a_frag, b_frag, c_frag_fp16);  // FP16 accumulation
            wmma::mma_sync(c_frag_fp32, a_frag, b_frag, c_frag_fp32);  // FP32 accumulation
        }
    }

    // Store the outputs
    wmma::store_matrix_sync(c_fp16 + warpM * 16 * N + warpN * 16, c_frag_fp16, N, wmma::mem_row_major);
    wmma::store_matrix_sync(c_fp32 + warpM * 16 * N + warpN * 16, c_frag_fp32, N, wmma::mem_row_major);
}

// Function to check if current GPU supports FP16 accumulation
bool checkTensorCoreFP16Support() {
    hipDeviceProp_t prop;
    hipGetDeviceProperties(&prop, 0);
    
    std::cout << "GPU: " << prop.name << std::endl;
    std::cout << "Compute Capability: " << prop.major << "." << prop.minor << std::endl;
    
    // Check for Tensor Core support
    if (prop.major >= 7) {
        std::cout << "✅ Tensor Cores supported" << std::endl;
        
        // Check for FP16 accumulation support (Ampere+)
        if (prop.major >= 8) {
            std::cout << "✅ FP16 accumulation supported" << std::endl;
            return true;
        } else {
            std::cout << "❌ FP16 accumulation NOT supported (only FP32 accumulation)" << std::endl;
            std::cout << "   GPU generation: " << (prop.major == 7 ? (prop.minor == 0 ? "Volta" : "Turing") : "Unknown") << std::endl;
            return false;
        }
    } else {
        std::cout << "❌ Tensor Cores NOT supported" << std::endl;
        return false;
    }
}

// Initialize test matrices
void initializeMatrices(std::vector<half>& a, std::vector<half>& b, int M, int N, int K) {
    // Initialize with small random values to avoid overflow in FP16
    for (int i = 0; i < M * K; i++) {
        a[i] = __float2half(((float)rand() / RAND_MAX - 0.5f) * 0.1f);
    }
    for (int i = 0; i < K * N; i++) {
        b[i] = __float2half(((float)rand() / RAND_MAX - 0.5f) * 0.1f);
    }
}

// Compare FP16 and FP32 accumulation results
void compareResults(const std::vector<half>& c_fp16, const std::vector<float>& c_fp32, 
                   int M, int N) {
    double maxAbsDiff = 0.0;
    double avgAbsDiff = 0.0;
    double maxRelDiff = 0.0;
    
    for (int i = 0; i < M * N; i++) {
        float val_fp16 = __half2float(c_fp16[i]);
        float val_fp32 = c_fp32[i];
        
        double absDiff = std::abs(val_fp16 - val_fp32);
        double relDiff = val_fp32 != 0.0f ? absDiff / std::abs(val_fp32) : 0.0;
        
        maxAbsDiff = std::max(maxAbsDiff, absDiff);
        avgAbsDiff += absDiff;
        maxRelDiff = std::max(maxRelDiff, relDiff);
    }
    
    avgAbsDiff /= (M * N);
    
    std::cout << "\n📊 Precision Analysis:" << std::endl;
    std::cout << "   Max Absolute Difference: " << maxAbsDiff << std::endl;
    std::cout << "   Avg Absolute Difference: " << avgAbsDiff << std::endl;
    std::cout << "   Max Relative Difference: " << maxRelDiff * 100 << "%" << std::endl;
    
    // Analyze the precision loss
    if (maxRelDiff > 0.01) {  // 1% threshold
        std::cout << "⚠️  Significant precision loss detected with FP16 accumulation" << std::endl;
    } else if (maxRelDiff > 0.001) {  // 0.1% threshold
        std::cout << "⚠️  Moderate precision loss detected with FP16 accumulation" << std::endl;
    } else {
        std::cout << "✅ Minimal precision loss with FP16 accumulation" << std::endl;
    }
}

int main() {
    std::cout << "=== Tensor Core FP16 Accumulation Test ===" << std::endl;
    
    // Check GPU capabilities
    if (!checkTensorCoreFP16Support()) {
        std::cout << "\n❌ This GPU does not support FP16 accumulation in Tensor Cores." << std::endl;
        std::cout << "   FP16 accumulation requires Ampere architecture or newer (Compute Capability >= 8.0)" << std::endl;
        return 1;
    }
    
    // Matrix dimensions (must be multiples of 16 for WMMA)
    const int M = 64, N = 64, K = 64;
    
    std::cout << "\n🔬 Testing matrix multiplication: " << M << "x" << K << " * " << K << "x" << N << std::endl;
    
    // Host matrices
    std::vector<half> h_a(M * K);
    std::vector<half> h_b(K * N);
    std::vector<half> h_c_fp16(M * N);
    std::vector<float> h_c_fp32(M * N);
    
    // Initialize matrices
    srand(42);  // For reproducible results
    initializeMatrices(h_a, h_b, M, N, K);
    
    // Device matrices
    half *d_a, *d_b, *d_c_fp16;
    float *d_c_fp32;
    
    hipMalloc(&d_a, M * K * sizeof(half));
    hipMalloc(&d_b, K * N * sizeof(half));
    hipMalloc(&d_c_fp16, M * N * sizeof(half));
    hipMalloc(&d_c_fp32, M * N * sizeof(float));
    
    // Copy to device
    hipMemcpy(d_a, h_a.data(), M * K * sizeof(half), hipMemcpyHostToDevice);
    hipMemcpy(d_b, h_b.data(), K * N * sizeof(half), hipMemcpyHostToDevice);
    
    // Launch kernel
    dim3 blockDim(32, 32);
    dim3 gridDim((M + 15) / 16, (N + 15) / 16);
    
    std::cout << "🚀 Launching Tensor Core computation..." << std::endl;
    
    tensorcore_fp16_accumulation_test<<<gridDim, blockDim>>>(
        d_a, d_b, d_c_fp16, d_c_fp32, M, N, K);
    
    hipError_t err = hipGetLastError();
    if (err != hipSuccess) {
        std::cerr << "CUDA Error: " << hipGetErrorString(err) << std::endl;
        return 1;
    }
    
    hipDeviceSynchronize();
    
    // Copy results back
    hipMemcpy(h_c_fp16.data(), d_c_fp16, M * N * sizeof(half), hipMemcpyDeviceToHost);
    hipMemcpy(h_c_fp32.data(), d_c_fp32, M * N * sizeof(float), hipMemcpyDeviceToHost);
    
    // Compare results
    compareResults(h_c_fp16, h_c_fp32, M, N);
    
    std::cout << "\n✅ Test completed successfully!" << std::endl;
    std::cout << "   Both FP16 and FP32 accumulation modes executed on Tensor Cores." << std::endl;
    
    // Cleanup
    hipFree(d_a);
    hipFree(d_b);
    hipFree(d_c_fp16);
    hipFree(d_c_fp32);
    
    return 0;
}